#include "hip/hip_runtime.h"
#include <omp.h>

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX 10

__global__ void kernel(int tid)
{
    printf("\n I am omp thread %d\n", tid);
}

int main()
{
    int count = 0;
#pragma omp parallel num_threads(MAX)
    {
#pragma omp atomic
        count++;
        kernel<<<1, 1>>>(omp_get_thread_num());
    }
    hipDeviceSynchronize();
    printf_s("Number of threads: %d\n", count);
}